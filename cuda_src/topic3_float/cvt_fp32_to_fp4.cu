
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_fp4.h>

using namespace std;

__global__ void execute_on_gpu(void) {
    // unsigned cal_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned x = 0U | (0x7dU << 23) | (0x600000U - 1U);
    // unsigned x = 0U | (0x7dU << 23);
    for (unsigned x = 0x7dU << 23; x <= 0xffffffff; x++) {
        __hip_fp4_storage_t res = __hip_cvt_float_to_fp4(*((float*)&x), __HIP_E2M1, hipRoundNearest);
        if (res != 0) {
            printf("%x, %f \n", x, *((float*)&x));
            printf("%d\n", (unsigned)res);
            assert(false);
        }
    }
}

int main(void) { 
	execute_on_gpu<<<1, 1>>>();
	hipDeviceSynchronize();
    return 0; 
}