
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp4.h>

using namespace std;

__global__ void test_fp4_to_fp16(void) {
    for (uint8_t src = 0; src < (1 << 4); src++) { 
        __half_raw res = __hip_cvt_fp4_to_halfraw(*((__hip_fp4_storage_t*)&src), __HIP_E2M1);
        printf("res = 0x%x, src = 0x%x, abs = 0x%x\n", *((uint16_t*)&res), (uint16_t)src, *((uint16_t*)&res) & 0x7fffU);
    }
}

int main(void) { 
	test_fp4_to_fp16<<<1, 1>>>();
	hipDeviceSynchronize();
    return 0; 
}